#include "hip/hip_runtime.h"
/******************************************************************************
devices.cu
Copyright (C) 2016  {fullname}

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License along
with this program; if not, write to the Free Software Foundation, Inc.,
51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.

Correspondence concerning RMSynth_GPU should be addressed to: 
sarrvesh.ss@gmail.com

******************************************************************************/
extern "C" {
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include "structures.h"
#include "constants.h"
#include "devices.h"
#include "fileaccess.h"
__global__ void computeQ(float *d_qImageArray, float *d_uImageArray, 
                         float *d_qPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2);
__global__ void computeU(float *d_qImageArray, float *d_uImageArray, 
                         float *d_uPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2);
__global__ void initializeQUP(float *d_qPhi, float *d_uPhi, 
                              float *d_pPhi, int nPhi);
__global__ void computeP(float *d_qPhi, float *d_uPhi, float *d_pPhi);
__global__ void computeQUP(float *d_qImageArray, float *d_uImageArray, int nChan,
                           float K, float *d_qPhi, float *d_uPhi, float *d_pPhi,  
                           float *d_phiAxis, int nPhi, float *d_lambdaDiff2);
void getGpuAllocForRMSynth(int *blockSize, int *threadSize, int nPhi,
                           struct deviceInfoList selectedDeviceInfo);
}

/*************************************************************
*
* Check if CUDA ERROR flag has been set. If raised, print 
*   error message to stdout and exit.
*
*************************************************************/
extern "C"
void checkCudaError() {
    hipError_t errorID = hipGetLastError();
    if(errorID != hipSuccess) {
        printf("\nERROR: %s", hipGetErrorString(errorID));
        exit(FAILURE);
    }
}

/*************************************************************
*
* Check for valid CUDA supported devices. If detected, 
*  print useful device information
*
*************************************************************/
extern "C"
struct deviceInfoList * getDeviceInformation(int *nDevices) {
    int dev;
    int deviceCount = NO_DEVICE;
    struct hipDeviceProp_t deviceProp;
    struct deviceInfoList *gpuList;
    
    /* Check for valid devices */
    hipDeviceReset();
    hipGetDeviceCount(&deviceCount);
    checkCudaError();
    if(deviceCount == NO_DEVICE) {
        printf("\nError: Could not detect CUDA supported GPU(s)\n\n");
        exit(FAILURE);
    }
    printf("\nINFO: Detected %d CUDA-supported GPU(s)\n", deviceCount);
    *nDevices = deviceCount;

    /* Store useful information about each GPU in a structure array */
    gpuList = (deviceInfoList *)malloc(deviceCount * 
      sizeof(struct deviceInfoList));
    for(dev=0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        checkCudaError();
        gpuList[dev].deviceID    = dev;
        gpuList[dev].globalMem   = deviceProp.totalGlobalMem;
        gpuList[dev].constantMem = deviceProp.totalConstMem;
        gpuList[dev].sharedMemPerBlock = deviceProp.sharedMemPerBlock;
        gpuList[dev].maxThreadPerMP = deviceProp.maxThreadsPerMultiProcessor;
        gpuList[dev].maxThreadPerBlock = deviceProp.maxThreadsPerBlock;
        gpuList[dev].threadBlockSize[0] = deviceProp.maxThreadsDim[0];
        gpuList[dev].threadBlockSize[1] = deviceProp.maxThreadsDim[1];
        gpuList[dev].threadBlockSize[2] = deviceProp.maxThreadsDim[2];
        gpuList[dev].warpSize           = deviceProp.warpSize;
        gpuList[dev].nSM                = deviceProp.multiProcessorCount;
        /* Print device info */
        printf("\nDevice %d: %s (version: %d.%d)", dev, deviceProp.name, 
                                                        deviceProp.major, 
                                                        deviceProp.minor);
        printf("\n\tGlobal memory: %f MB", gpuList[dev].globalMem/MEGA);
        printf("\n\tShared memory: %f kB", gpuList[dev].sharedMemPerBlock/KILO);
        printf("\n\tMax threads per block: %d", gpuList[dev].maxThreadPerBlock);
        printf("\n\tMax threads per MP: %d", gpuList[dev].maxThreadPerMP);
        printf("\n\tProcessor count: %d", deviceProp.multiProcessorCount);
        printf("\n\tMax thread dim: (%d, %d, %d)", deviceProp.maxThreadsDim[0], 
                                                   deviceProp.maxThreadsDim[1], 
                                                   deviceProp.maxThreadsDim[2]);
    }
    printf("\n");
    return(gpuList);
}

/*************************************************************
*
* Select the best GPU device
*
*************************************************************/
extern "C"
int getBestDevice(struct deviceInfoList *gpuList, int nDevices) {
    int dev=0;
    int i, maxMem;
    if(nDevices == 1) { dev = 0; }
    else {
        maxMem = gpuList[dev].globalMem;
        for(i=1; i<nDevices; i++) {
            if(maxMem < gpuList[i].globalMem) { 
                maxMem = gpuList[i].globalMem;
                dev = i;
            }
            else { continue; }
        }
    }
    return dev;
}

/*************************************************************
*
* Copy GPU device information of selectedDevice from gpuList 
*  to selectedDevice
*
*************************************************************/
extern "C"
struct deviceInfoList copySelectedDeviceInfo(struct deviceInfoList *gpuList, 
                                             int selectedDevice) {
    int i = selectedDevice;
    struct deviceInfoList selectedDeviceInfo;
    selectedDeviceInfo.deviceID           = gpuList[i].deviceID;
    selectedDeviceInfo.globalMem          = gpuList[i].globalMem;
    selectedDeviceInfo.constantMem        = gpuList[i].constantMem;
    selectedDeviceInfo.sharedMemPerBlock  = gpuList[i].sharedMemPerBlock;
    selectedDeviceInfo.maxThreadPerMP     = gpuList[i].maxThreadPerMP;
    selectedDeviceInfo.maxThreadPerBlock  = gpuList[i].maxThreadPerBlock;
    selectedDeviceInfo.threadBlockSize[0] = gpuList[i].threadBlockSize[0];
    selectedDeviceInfo.threadBlockSize[1] = gpuList[i].threadBlockSize[1];
    selectedDeviceInfo.threadBlockSize[2] = gpuList[i].threadBlockSize[2];
    selectedDeviceInfo.warpSize           = gpuList[i].warpSize;
    selectedDeviceInfo.nSM                = gpuList[i].nSM;
    return selectedDeviceInfo;
}

/*************************************************************
*
* GPU accelerated RM Synthesis function
*
*************************************************************/
extern "C"
int doRMSynthesis(struct optionsList *inOptions, struct parList *params,
                  struct deviceInfoList selectedDeviceInfo) {
    int i, j, k; 
    float *lambdaDiff2, *d_lambdaDiff2;
    size_t size;
    float *qImageArray, *uImageArray;
    float *d_qImageArray, *d_uImageArray;
    float *d_qPhi, *d_uPhi, *d_pPhi;
    float *d_phiAxis;
    int calcThreadSize, calcBlockSize;
    hipEvent_t startEvent, stopEvent;
    hipEvent_t tStart, tStop;
    float millisec = 0.;
    long fPixel[params->qAxisLen3];
    int fitsStatus = 0;
    
    /* Initialize CUDA events to measure time */
    hipEventCreate(&startEvent); hipEventCreate(&tStart);
    hipEventCreate(&stopEvent);  hipEventCreate(&tStop);

    /* Compute \lambda^2 - \lambda^2_0 once */
    lambdaDiff2 = (float *)calloc(params->qAxisLen3, sizeof(lambdaDiff2));
    if(lambdaDiff2 == NULL) {
        printf("ERROR: Mem alloc failed for lambdaDiff2\n\n");
        return(FAILURE);
    }
    for(i=0;i<params->qAxisLen3;i++)
        lambdaDiff2[i] = 2.0*(params->lambda2[i]-params->lambda20);
    
    /* Allocate input arrays on CPU */
    qImageArray = (float *)calloc(params->qAxisLen3, sizeof(qImageArray));
    uImageArray = (float *)calloc(params->qAxisLen3, sizeof(uImageArray));
    /* Allocate and initialize input arrays on GPU */
    size = sizeof(d_qImageArray)*params->qAxisLen3;
    hipMalloc(&d_qImageArray, size);
    hipMalloc(&d_uImageArray, size);
    hipMalloc(&d_lambdaDiff2, size);
    hipMemcpy(d_lambdaDiff2, lambdaDiff2, size, hipMemcpyHostToDevice);
    /* Allocate and initialize output arrays on GPU */
    size = sizeof(d_qPhi)*inOptions->nPhi;
    hipMalloc(&d_qPhi, size); 
    hipMalloc(&d_uPhi, size);
    hipMalloc(&d_pPhi, size);
    hipMalloc(&d_phiAxis, size);
    hipMemcpy(d_phiAxis, params->phiAxis, size, hipMemcpyHostToDevice);
    checkCudaError();

    /* Start the clock */
    hipEventRecord(startEvent);

    /* Determine what the appropriate block and grid sizes are */
    calcThreadSize = selectedDeviceInfo.warpSize;
    calcBlockSize  = inOptions->nPhi/calcThreadSize + 1;
    printf("INFO: Launching %d blocks each with %d threads\n", 
            calcBlockSize, calcThreadSize);

    /* Process each line of sight individually */
    size = sizeof(d_qImageArray)*params->qAxisLen3;
    for(i=1; i<=params->qAxisLen1; i++) {
        fPixel[1] = i;
        for(j=1; j<=params->qAxisLen2; j++) {
            fPixel[0] = j;
    
            /* Read this line of sight from Q and U array */
            hipEventRecord(tStart);
            for(k=1; k<=params->qAxisLen3; k++) {
               fPixel[2] = k;
               fits_read_pix(params->qFile, TFLOAT, fPixel, 1, NULL, 
                             &(qImageArray[k-1]), NULL, &fitsStatus);
               fits_read_pix(params->uFile, TFLOAT, fPixel, 1, NULL,
                             &(uImageArray[k-1]), NULL, &fitsStatus);
               checkFitsError(fitsStatus);
            }
            hipEventRecord(tStop);
            hipEventSynchronize(tStop);
            hipEventElapsedTime(&millisec, tStart, tStop);
            printf("INFO: %0.2f ms to read fits data\n", millisec);
            
            /* Move Q(lambda) and U(lambda) to device */
            hipEventRecord(tStart);
            hipMemcpy(d_qImageArray, qImageArray, size,
                       hipMemcpyHostToDevice);
            hipMemcpy(d_uImageArray, uImageArray, size,
                       hipMemcpyHostToDevice);
            hipEventRecord(tStop);
            hipEventSynchronize(tStop);
            hipEventElapsedTime(&millisec, tStart, tStop);
            printf("INFO: %0.2f ms to move data to gpu\n", millisec);
            
            /* Launch kernels to compute Q(\phi), U(\phi), and P(\phi) */
            hipEventRecord(tStart);
            computeQUP<<<calcBlockSize, calcThreadSize>>>(d_qImageArray,
                         d_uImageArray, params->qAxisLen3, params->K, d_qPhi, 
                         d_uPhi, d_pPhi, d_phiAxis, inOptions->nPhi, d_lambdaDiff2);
            hipEventRecord(tStop);
            hipEventSynchronize(tStop);
            hipEventElapsedTime(&millisec, tStart, tStop);
            printf("INFO: %0.2f ms to process data\n", millisec);

            /* Move Q(\phi), U(\phi) and P(\phi) to host */
            break; /* THIS IS UNWANTED. ONLY FOR TESTING PURPOSE */
        }
        break;
    }
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&millisec, startEvent, stopEvent);
    printf("INFO: Time to process the cubes: %0.2f s.\n", millisec);
    
    return(SUCCESS);
}

/*************************************************************
*
* Device code to compute Q(\phi)
*
*************************************************************/
extern "C"
__global__ void computeQUP(float *d_qImageArray, float *d_uImageArray, int nChan, 
                           float K, float *d_qPhi, float *d_uPhi, float *d_pPhi, 
                           float *d_phiAxis, int nPhi, float *d_lambdaDiff2) {
    int i;
    const int index   = blockIdx.x*blockDim.x + threadIdx.x;
    const float myphi = d_phiAxis[index];
    float qPhi, uPhi, pPhi;
    qPhi = 0; uPhi = 0;

    if(index < nPhi) {
        for(i=0; i<nChan; i++) {
            qPhi += d_qImageArray[i]*cosf(myphi*d_lambdaDiff2[i]) + 
                    d_uImageArray[i]*sinf(myphi*d_lambdaDiff2[i]);
            uPhi += d_uImageArray[i]*cosf(myphi*d_lambdaDiff2[i]) -
                    d_qImageArray[i]*cosf(myphi*d_lambdaDiff2[i]);
        }
        pPhi = sqrt(qPhi*qPhi + uPhi*uPhi);

        d_qPhi[index] = K*qPhi;
        d_uPhi[index] = K*uPhi;
        d_pPhi[index] = K*pPhi;
    }
}

/*************************************************************
*
* Device code to compute Q(\phi)
*
*************************************************************/
extern "C"
__global__ void computeQ(float *d_qImageArray, float *d_uImageArray, 
                         float *d_qPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2) {
    int i;
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int offset = index*nElements;
    const float thisPhi = d_phiAxis[index];    
    const float thisCos = cosf(2*thisPhi*dlambda2);
    const float thisSin = sinf(2*thisPhi*dlambda2);

    if(index < nPhi) {
        /* For each element in Q, compute Q(thisPhi) and add it to Q(phi) */
        for(i=0; i<nElements; i++)
            d_qPhi[offset+i] += d_qImageArray[i]*thisCos - 
                                d_uImageArray[i]*thisSin;
    }
}

/*************************************************************
*
* Device code to compute U(\phi)
*
*************************************************************/
extern "C"
__global__ void computeU(float *d_qImageArray, float *d_uImageArray, 
                         float *d_uPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2) {
    int i;
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int offset = index*nElements;
    const float thisPhi = d_phiAxis[index];
    const float thisCos = cosf(2*thisPhi*dlambda2);
    const float thisSin = sinf(2*thisPhi*dlambda2);

    if(index < nPhi) {
        /* For each element in U, compute U(thisPhi) and add it to U(phi) */
        for(i=0; i<nElements; i++) 
            d_uPhi[offset+i] += d_uImageArray[i]*thisCos - 
                                d_qImageArray[i]*thisSin;
    }
}

/*************************************************************
*
* Initialize Q(\phi) and U(\phi)
*
*************************************************************/
extern "C"
__global__ void initializeQUP(float *d_qPhi, float *d_uPhi, 
                              float *d_pPhi, int nPhi) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < nPhi) {
        d_qPhi[index] = 0.0;
        d_uPhi[index] = 0.0;
        d_pPhi[index] = 0.0;
    }
}

/*************************************************************
*
* Estimate the optimal number of block and thread size 
*  for RM Synthesis.
*
*************************************************************/
extern "C"
void getGpuAllocForRMSynth(int *blockSize, int *threadSize, int nPhi,
                           struct deviceInfoList selectedDeviceInfo) {
    *threadSize = selectedDeviceInfo.warpSize;
    if(!(*blockSize = nPhi/(*threadSize))) { *blockSize = 1; }
}

/*************************************************************
*
* Estimate the optimal number of block, thread and memory to
*  compute P from Q and U images/cubes.
*
*************************************************************/
extern "C"
void getGpuAllocForP(int *blockSize, int *threadSize, long *nFrames, 
                     int nImRows, int nRowElements, 
                     struct deviceInfoList selectedDeviceInfo) {
    long totalThreads;

    /* How many phi frames can be stored in gpu at a time */
    *nFrames = (int)(selectedDeviceInfo.globalMem % 
              (3*nImRows*nRowElements*sizeof(float)));
    
    /* Determine the thread and block size */
    totalThreads = *nFrames;
    if(totalThreads <= selectedDeviceInfo.maxThreadPerBlock) {
        *threadSize = totalThreads;
        *blockSize = 1;
    }
    else {
        *threadSize = selectedDeviceInfo.maxThreadPerBlock;
        *blockSize = (totalThreads % selectedDeviceInfo.maxThreadPerBlock) + 1;
    }
}

extern "C" {
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include "structures.h"
#include "constants.h"
#include "devices.h"
}

/*************************************************************
*
* Check for valid CUDA supported devices. If detected, 
*  print useful device information
*
*************************************************************/
extern "C"
struct deviceInfoList * getDeviceInformation(int *nDevices) {
    int dev;
    hipError_t errorID;
    int deviceCount = NO_DEVICE;
    struct hipDeviceProp_t deviceProp;
    struct deviceInfoList *gpuList;
    
    /* Check for valid devices */
    hipDeviceReset();
    errorID = hipGetDeviceCount(&deviceCount);
    if(errorID != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n%s", (int)errorID, 
               hipGetErrorString(errorID));
        exit(FAILURE);
    }
    if(deviceCount == NO_DEVICE) {
        printf("\nError: Could not detect CUDA supported GPU(s)\n\n");
        exit(FAILURE);
    }
    printf("\nINFO: Detected %d CUDA-supported GPU(s)\n", deviceCount);
    *nDevices = deviceCount;

    /* Store useful information about each GPU in a structure array */
    gpuList = (deviceInfoList *)malloc(deviceCount * sizeof(struct deviceInfoList));
    for(dev=0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %d: %s", dev, deviceProp.name);
        gpuList[dev].deviceID    = dev;
        gpuList[dev].globalMem   = deviceProp.totalGlobalMem;
        gpuList[dev].constantMem = deviceProp.totalConstMem;
        gpuList[dev].sharedMemPerBlock = deviceProp.sharedMemPerBlock;
        gpuList[dev].maxThreadPerMP = deviceProp.maxThreadsPerMultiProcessor;
        gpuList[dev].maxThreadPerBlock = deviceProp.maxThreadsPerBlock;
        gpuList[dev].threadBlockSize[0] = deviceProp.maxThreadsDim[0];
        gpuList[dev].threadBlockSize[1] = deviceProp.maxThreadsDim[1];
        gpuList[dev].threadBlockSize[2] = deviceProp.maxThreadsDim[2];
        printf("\n%d", deviceProp.multiProcessorCount);
        printf("\n%d %d %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("\n%ld", deviceProp.maxThreadsDim[0]*deviceProp.maxThreadsDim[1]*deviceProp.maxThreadsDim[2]);
    }
    printf("\n");
    return(gpuList);
}

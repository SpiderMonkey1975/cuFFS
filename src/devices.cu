#include "hip/hip_runtime.h"
/******************************************************************************
devices.cu
Copyright (C) 2016  {fullname}

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License along
with this program; if not, write to the Free Software Foundation, Inc.,
51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.

Correspondence concerning RMSynth_GPU should be addressed to: 
sarrvesh.ss@gmail.com

******************************************************************************/
extern "C" {
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include "structures.h"
#include "constants.h"
#include "devices.h"
#include "fileaccess.h"
__global__ void computeQ(float *d_qImageArray, float *d_uImageArray, 
                         float *d_qPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2);
__global__ void computeU(float *d_qImageArray, float *d_uImageArray, 
                         float *d_uPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2);
__global__ void initializeQU(float *d_array, int nElements, int nPhi);
__global__ void computeP(float *d_qPhi, float *d_uPhi, float *d_pPhi);
void getGpuAllocForRMSynth(int *blockSize, int *threadSize, int nPhi,
                           struct deviceInfoList selectedDeviceInfo);
}

/*************************************************************
*
* Check if CUDA ERROR flag has been set. If raised, print 
*   error message to stdout and exit.
*
*************************************************************/
extern "C"
void checkCudaError() {
    hipError_t errorID = hipGetLastError();
    if(errorID != hipSuccess) {
        printf("\nERROR: %s", hipGetErrorString(errorID));
        exit(FAILURE);
    }
}

/*************************************************************
*
* Check for valid CUDA supported devices. If detected, 
*  print useful device information
*
*************************************************************/
extern "C"
struct deviceInfoList * getDeviceInformation(int *nDevices) {
    int dev;
    int deviceCount = NO_DEVICE;
    struct hipDeviceProp_t deviceProp;
    struct deviceInfoList *gpuList;
    
    /* Check for valid devices */
    hipDeviceReset();
    hipGetDeviceCount(&deviceCount);
    checkCudaError();
    if(deviceCount == NO_DEVICE) {
        printf("\nError: Could not detect CUDA supported GPU(s)\n\n");
        exit(FAILURE);
    }
    printf("\nINFO: Detected %d CUDA-supported GPU(s)\n", deviceCount);
    *nDevices = deviceCount;

    /* Store useful information about each GPU in a structure array */
    gpuList = (deviceInfoList *)malloc(deviceCount * 
      sizeof(struct deviceInfoList));
    for(dev=0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        checkCudaError();
        gpuList[dev].deviceID    = dev;
        gpuList[dev].globalMem   = deviceProp.totalGlobalMem;
        gpuList[dev].constantMem = deviceProp.totalConstMem;
        gpuList[dev].sharedMemPerBlock = deviceProp.sharedMemPerBlock;
        gpuList[dev].maxThreadPerMP = deviceProp.maxThreadsPerMultiProcessor;
        gpuList[dev].maxThreadPerBlock = deviceProp.maxThreadsPerBlock;
        gpuList[dev].threadBlockSize[0] = deviceProp.maxThreadsDim[0];
        gpuList[dev].threadBlockSize[1] = deviceProp.maxThreadsDim[1];
        gpuList[dev].threadBlockSize[2] = deviceProp.maxThreadsDim[2];
        gpuList[dev].warpSize           = deviceProp.warpSize;
        gpuList[dev].nSM                = deviceProp.multiProcessorCount;
        /* Print device info */
        printf("\nDevice %d: %s (version: %d.%d)", dev, deviceProp.name, 
                                                        deviceProp.major, 
                                                        deviceProp.minor);
        printf("\n\tGlobal memory: %f MB", gpuList[dev].globalMem/MEGA);
        printf("\n\tShared memory: %f kB", gpuList[dev].sharedMemPerBlock/KILO);
        printf("\n\tMax threads per block: %d", gpuList[dev].maxThreadPerBlock);
        printf("\n\tMax threads per MP: %d", gpuList[dev].maxThreadPerMP);
        printf("\n\tProcessor count: %d", deviceProp.multiProcessorCount);
        printf("\n\tMax thread dim: (%d, %d, %d)", deviceProp.maxThreadsDim[0], 
                                                   deviceProp.maxThreadsDim[1], 
                                                   deviceProp.maxThreadsDim[2]);
    }
    printf("\n");
    return(gpuList);
}

/*************************************************************
*
* Select the best GPU device
*
*************************************************************/
extern "C"
int getBestDevice(struct deviceInfoList *gpuList, int nDevices) {
    int dev=0;
    int i, maxMem;
    if(nDevices == 1) { dev = 0; }
    else {
        maxMem = gpuList[dev].globalMem;
        for(i=1; i<nDevices; i++) {
            if(maxMem < gpuList[i].globalMem) { 
                maxMem = gpuList[i].globalMem;
                dev = i;
            }
            else { continue; }
        }
    }
    return dev;
}

/*************************************************************
*
* Copy GPU device information of selectedDevice from gpuList 
*  to selectedDevice
*
*************************************************************/
extern "C"
struct deviceInfoList copySelectedDeviceInfo(struct deviceInfoList *gpuList, 
                                             int selectedDevice) {
    int i = selectedDevice;
    struct deviceInfoList selectedDeviceInfo;
    selectedDeviceInfo.deviceID           = gpuList[i].deviceID;
    selectedDeviceInfo.globalMem          = gpuList[i].globalMem;
    selectedDeviceInfo.constantMem        = gpuList[i].constantMem;
    selectedDeviceInfo.sharedMemPerBlock  = gpuList[i].sharedMemPerBlock;
    selectedDeviceInfo.maxThreadPerMP     = gpuList[i].maxThreadPerMP;
    selectedDeviceInfo.maxThreadPerBlock  = gpuList[i].maxThreadPerBlock;
    selectedDeviceInfo.threadBlockSize[0] = gpuList[i].threadBlockSize[0];
    selectedDeviceInfo.threadBlockSize[1] = gpuList[i].threadBlockSize[1];
    selectedDeviceInfo.threadBlockSize[2] = gpuList[i].threadBlockSize[2];
    selectedDeviceInfo.warpSize           = gpuList[i].warpSize;
    selectedDeviceInfo.nSM                = gpuList[i].nSM;
    return selectedDeviceInfo;
}

/*************************************************************
*
* GPU accelerated RM Synthesis function
*
*************************************************************/
extern "C"
int doRMSynthesis(struct optionsList *inOptions, struct parList *params,
                  struct deviceInfoList selectedDeviceInfo) {
    long unsigned int losSize, nLOS; 

    /* Check how many lines of sight will fit into gpu memory */
    losSize = sizeof(float)*params->qAxisLen3 +
              sizeof(float)*inOptions->nPhi;
    nLosAtOnce = selectedDeviceInfo->globalMem/losSize;
    nLOS = params->qAxisLen1 * params->qAxisLen2;
    
    printf("INFO: Will process %d/%d lines of sight at once.\n",
           nLosAtOnce, nLOS);
    
    return(SUCCESS);
}

/*************************************************************
*
* Device code to compute Q(\phi)
*
*************************************************************/
extern "C"
__global__ void computeQ(float *d_qImageArray, float *d_uImageArray, 
                         float *d_qPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2) {
    int i;
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int offset = index*nElements;
    const float thisPhi = d_phiAxis[index];    
    const float thisCos = cosf(2*thisPhi*dlambda2);
    const float thisSin = sinf(2*thisPhi*dlambda2);

    if(index < nPhi) {
        /* For each element in Q, compute Q(thisPhi) and add it to Q(phi) */
        for(i=0; i<nElements; i++)
            d_qPhi[offset+i] += d_qImageArray[i]*thisCos - 
                                d_uImageArray[i]*thisSin;
    }
}

/*************************************************************
*
* Device code to compute U(\phi)
*
*************************************************************/
extern "C"
__global__ void computeU(float *d_qImageArray, float *d_uImageArray, 
                         float *d_uPhi, float *d_phiAxis, int nPhi, 
                         int nElements, float dlambda2) {
    int i;
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int offset = index*nElements;
    const float thisPhi = d_phiAxis[index];
    const float thisCos = cosf(2*thisPhi*dlambda2);
    const float thisSin = sinf(2*thisPhi*dlambda2);

    if(index < nPhi) {
        /* For each element in U, compute U(thisPhi) and add it to U(phi) */
        for(i=0; i<nElements; i++) 
            d_uPhi[offset+i] += d_uImageArray[i]*thisCos - 
                                d_qImageArray[i]*thisSin;
    }
}

/*************************************************************
*
* Initialize Q(\phi) and U(\phi)
*
*************************************************************/
extern "C"
__global__ void initializeQU(float *d_array, int nElements, int nPhi) {
    int i;
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < nPhi) {
        for(i=0; i<nElements; i++)
            d_array[index*nElements+i] = 0.0;
    }
}

/*************************************************************
*
* Estimate the optimal number of block and thread size 
*  for RM Synthesis.
*
*************************************************************/
extern "C"
void getGpuAllocForRMSynth(int *blockSize, int *threadSize, int nPhi,
                           struct deviceInfoList selectedDeviceInfo) {
    *blockSize = selectedDeviceInfo.maxThreadPerBlock;
    if(nPhi < *blockSize)
        *threadSize = 1;
    else
        *threadSize = (int) nPhi/(*blockSize) + 1;
}

/*************************************************************
*
* Estimate the optimal number of block, thread and memory to
*  compute P from Q and U images/cubes.
*
*************************************************************/
extern "C"
void getGpuAllocForP(int *blockSize, int *threadSize, long *nFrames, 
                     int nImRows, int nRowElements, 
                     struct deviceInfoList selectedDeviceInfo) {
    long totalThreads;

    /* How many phi frames can be stored in gpu at a time */
    *nFrames = (int)(selectedDeviceInfo.globalMem % 
              (3*nImRows*nRowElements*sizeof(float)));
    
    /* Determine the thread and block size */
    totalThreads = *nFrames;
    if(totalThreads <= selectedDeviceInfo.maxThreadPerBlock) {
        *threadSize = totalThreads;
        *blockSize = 1;
    }
    else {
        *threadSize = selectedDeviceInfo.maxThreadPerBlock;
        *blockSize = (totalThreads % selectedDeviceInfo.maxThreadPerBlock) + 1;
    }
}
